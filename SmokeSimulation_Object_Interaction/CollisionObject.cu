#include "CollisionObject.cuh"

int* CollisionObject::d_calcCollision = nullptr;
int* CollisionObject::d_drawCollision = nullptr;
int* CollisionObject::d_ID = nullptr;

CollisionObject::CollisionObject(int N, float size, glm::vec3 oInfo[], float vel) {
	_N = N;
	_size = size;
	_start_pos = oInfo[0];
	_curr_pos = oInfo[0];
	_prev_pos = oInfo[0];
	_dir = oInfo[1];
	_vel = vel;
}

CollisionObject::~CollisionObject() {

}

void CollisionObject::initialize_memory(int N) {
	hipMalloc((void**)&d_calcCollision, (N + 2) * (N + 2) * (N + 2) * sizeof(int));
	hipMalloc((void**)&d_drawCollision, N * N * N * sizeof(int));
	hipMalloc((void**)&d_ID, (N + 2) * (N + 2) * (N + 2) * sizeof(int));
}

void CollisionObject::finalize_memory() {
	hipFree(d_calcCollision);
	hipFree(d_drawCollision);
	hipFree(d_ID);
}